#include "hip/hip_runtime.h"
#include "ReSTIRDI.h"

struct Ray 
{
	float3 origin;
	float3 direction;
	__device__ Ray(const float3& m_origin, const float3& m_direction) : origin(m_origin), direction(m_direction) {}
};

enum Refl_t { DIFF, SPEC };

struct Sphere 
{
	float radius;
	float3 position, color;
	Refl_t refl;
	__device__ float Intersect(const Ray& ray) const {
		float3 op = position - ray.origin;
		float t, epsilon = 1e-4;
		float b = dot(op, ray.direction), det = b * b - dot(op, op) + radius * radius;
		if (det < 0) return 0;	else det = sqrtf(det);
		return (t = b - det) > epsilon ? t : ((t = b + det) > epsilon ? t : 0);
	}
};

struct PointLight 
{
	float3 position, emission;
};

__constant__ Sphere spheres[] = {//Scene: radius, position, color
	{1e5, {50, -100000, 0}, {.184,.929,.929}, DIFF},//ground
	{26.5, {22,26.5,42}, {.596,.596,.596}, SPEC},//Ball1
	{13, {75,13,82}, {.91,.29,.102}, DIFF},//Ball2
	{22,{87,22,24}, {.99,.405,.992}, DIFF},//Ball3
	{1e4,{50.0,-40.8,1e4-200}, {.4,.7,.9}, DIFF}//background
};

__constant__ PointLight lights[] = {// Scene: positioin, emission
	{{-50.0, 181.6 + 10.27, 176.6}, {5000.0, 9000.0, 5000.0}},
	{{0.0, 181.6 - 20.27, -126.6}, {1000.0, 5000.0, 7000.0}},
	{{50.0, 181.6 + 305.27, 146.6}, {6000.0, 5000.0, 5000.0}},
	{{80.0, 181.6 - 432.27, -156.6}, {5000.0, 3000.0, 5000.0}},
	{{-20.0, 181.6 + 59.27, 136.6}, {5000.0, 5000.0, 5000.0}},
	{{-10.0, 181.6 - 68.27, 156.6}, {5000.0, 9000.0, 5000.0}},
	{{10.0, 181.6 + 7.27, -106.6}, {5000.0, 5000.0, 9000.0}},
	{{30.0, 181.6 - 8.27, 126.6}, {1000.0, 5000.0, 5000.0}},
	{{60.0, 181.6 + 93.27, -146.6}, {6000.0, 5000.0, 5000.0}},
	{{100.0, 181.6 - 63.27, 166.6}, {5000.0, 3000.0, 5000.0}},
	{{-110.0, 181.6 + 74.27, -161.6}, {1e3, 6e3, 5e3}},
	{{-120.0, 181.6 + 75.27, 152.6}, {4e3, 6e3, 4e3}},
	{{-170.0, 181.6 - 21.27, -236.6}, {1e3, 6e3, 2e3}},
	{{-110.0, 181.6 + 12.27, 256.6}, {6e3, 8e3, 9e3}},
	{{-90.0, 181.6 - 41.27, 137.6}, {3e3, 2e3, 6e3}},
	{{-610.0, 181.6 + 312.27, -115.6}, {8e3, 4e3, 7e3}},
	{{-40.0, 181.6 + 123.27, -235.6}, {2e3, 5e3, 3e3}},
	{{-30.0, 181.6 + 145.27, 236.6}, {1e3, 1e3, 1e3}},
	{{-20.0, 181.6 - 93.27, -132.6}, {6e3, 3e3, 8e3}},
	{{-124.0, 181.6 + 12.27, 152.6}, {2e3, 2e3, 1e3}},
	{{-253.0, 181.6 + 54.27, -523.6}, {4e3, 8e3, 7e3}},
	{{340.0, 181.6 - 14.27, 123.6}, {6e3, 6e3, 1e3}},
	{{230.0, 181.6 + 136.27, -234.6}, {1e3, 3e3, 7e3}},
	{{560.0, 181.6 - 134.27, -45.6}, {7e3, 2e3, 3e3}},
	{{20.0, 181.6 + 34.27, 65.6}, {9e3, 9e3, 1e3}},
	{{80.0, 181.6 - 56.27, -513.6}, {2e3, 7e3, 2e3}},
	{{90.0, 181.6 + 23.27, 34.6}, {4e3, 2e3, 2e3}},
	{{245.0, 181.6 + 77.27, 74.6}, {5e3, 3e3, 6e3}},
	{{54.0, 181.6 + 88.27, -23.6}, {7e3, 8e3, 8e3}},
	{{86.0, 181.6 + 99.27, 45.6}, {3e3, 4e3, 9e3}},
	{{432.0, 181.6 - 14.27, -97.6}, {4e3, 6e3, 2e3}},
	{{744.0, 181.6 + 66.27, -28.6}, {7e3, 2e3, 4e3}},
	{{76.0, 181.6 + 34.27, -534.6}, {8e3, 2e3, 7e3}},
	{{43.0, 181.6 - 76.27, 532.6}, {1e3, 8e3, 3e3}},
	{{258.0, 181.6 + 85.27, -93.6}, {4e3, 7e3, 7e3}},
	{{224.0, 181.6 + 24.27, -76.6}, {8e3, 4e3, 3e3}},
	{{13.0, 181.6 + 45.27, 36.6}, {3e3, 7e3, 8e3}},
	{{64.0, 181.6 - 46.27, -34.6}, {6e3, 4e3, 5e3}},
	{{99.0, 181.6 - 32.27, -45.6}, {8e3, 6e3, 1e3}}
};

inline __host__ __device__ float Clamp(float x) 
{
	return x < 0 ? 0 : x > 1 ? 1 : x;
}

inline __host__ __device__ int ToInt(float x) 
{
	return int(pow(Clamp(x), 1 / 2.2) * 255 + .5);
}

inline __device__ bool Intersect(const Ray& ray, float& t, int& id) 
{
	float n = sizeof(spheres) / sizeof(Sphere), d, inf = t = 1e20;
	for (int i = int(n); i--;) if ((d = spheres[i].Intersect(ray)) && d < t)
	{
		t = d, id = i;
	}
	return t < inf;
}

__device__ void RISAndTemporalReuse(const int &frameNumber, const Ray& r, const int& M, hiprandState *randstate, int2 index, int pixelIndex, 
							Reservoir *previousReservoir, Reservoir *currentReservoir,
							bool temporalReuse, bool spatialReuse)
{
	float t; // distance to intersection
	int id = 0; // id of intersected object
	Ray ray = r;

	if (!Intersect(ray, t, id)) return;
	Sphere* obj = &spheres[id]; // the hit object
	float3 hitPoint = ray.origin + ray.direction * t;
	float3 normal = normalize(hitPoint - obj->position);
	float3 normalLocal = dot(normal, ray.direction) < 0 ? normal : normal * -1;

	// Perform specular reflection but not as iteration
	if (obj->refl == SPEC)
	{
		float3 reflectionDirection = ray.direction - normal * 2 * dot(normal, ray.direction);
		Ray reflectionRay = Ray(hitPoint + normal * 2e-2, reflectionDirection);
		if (!Intersect(reflectionRay, t, id)) return ;
		obj = &spheres[id]; // the hit object
		hitPoint = reflectionRay.origin + reflectionRay.direction * t;
		normal = normalize(hitPoint - obj->position);
		normalLocal = dot(normal, reflectionRay.direction) < 0 ? normal : normal * -1;
	}

	int lightsCount = sizeof(lights) / sizeof(PointLight);
	int RISSamples = lightsCount > M ? M : lightsCount;

	Reservoir reservoir;
	Reservoir visibility_reservoir;
	Reservoir temporal_reservoir;

	reservoir.normal = normalLocal;
	reservoir.depth = t;
	visibility_reservoir.normal = normalLocal;
	visibility_reservoir.depth = t;
	temporal_reservoir.normal = normalLocal;
	temporal_reservoir.depth = t;

	////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////// Candidate Generation ////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////////////////

	for (int i = 0; i < RISSamples; i++)
	{
		// Pick a random light from the scene to sample
		int randomLight = int(hiprand_uniform(randstate) * lightsCount);
		int lightToSample = randomLight > lightsCount - 1 ? lightsCount - 1 : randomLight;

		// Sample the light
		float distanceToLight = length(lights[lightToSample].position - hitPoint);
		float3 lightEmission = lights[lightToSample].emission;
		float3 lightDirection = normalize(lights[lightToSample].position - hitPoint);

		// Compute the Lambertian cosine
		float cosTheta = dot(normalLocal, lightDirection);
		if (cosTheta < 0) cosTheta = 0.;

		// Calculate the light attenuation
		float lightAttenuation = 1 / (distanceToLight * distanceToLight);

		// Compute the BRDF
		float3 BRDF = obj->color * INV_PI * cosTheta;

		// Compute the light intensity
		float3 lightIntensity = lightEmission * lightAttenuation;

		// Compute radiance
		float3 radiance = BRDF * lightIntensity;

		// Compute pHat
		float pHat = length(radiance);

		// MIS weight
		float MISWeight = 1. / RISSamples;

		// Contribution weight of light
		float contributionWeight = lightsCount; // The inverse PDF of sampling the light

		// The weight of the sample
		float weight = pHat * MISWeight * contributionWeight;

		// Add the sample to the reservoir
		reservoir.addSample(lightToSample, weight, randstate);
	}

	////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////  Visibility Pass ///////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////////////////

	// Check if the reservoir sample is visible
	int tempLight = reservoir.id;
	float v_distanceToLight = length(lights[tempLight].position - hitPoint);
	float3 v_lightEmission = lights[tempLight].emission;
	float3 v_lightDirection = normalize(lights[tempLight].position - hitPoint);
	float v_cosTheta = dot(normalLocal, v_lightDirection);
	if (v_cosTheta < 0) v_cosTheta = 0.;
	float v_visibility = 0;
	Ray v_shadowRay(hitPoint + normalLocal * 2e-2, v_lightDirection);
	if (!(Intersect(v_shadowRay, t, id) && t < length(lights[tempLight].position - hitPoint) - 2e-2))
	{
		v_visibility = 1;
	}
	float v_lightAttenuation = 1 / (v_distanceToLight * v_distanceToLight);
	float3 v_BRDF = obj->color * INV_PI * v_cosTheta;
	float3 v_lightIntensity = v_lightEmission * v_lightAttenuation;
	float3 v_radiance = v_BRDF * v_lightIntensity * v_visibility;
	float v_pHat = length(v_radiance);
	float v_weight = v_pHat > 0.0 ? (1. / v_pHat) * reservoir.wSum : 0.0;
	float v_ContributionWeight = v_pHat * v_weight * v_visibility;

	visibility_reservoir.addSample(tempLight, v_ContributionWeight, randstate);
	reservoir.id = visibility_reservoir.id;
	reservoir.wSum = visibility_reservoir.wSum;

	// Update the current reservoir
	currentReservoir[pixelIndex] = reservoir;

	////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////// Temporal Reuse Pass /////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////////////////

	if (temporalReuse && frameNumber > 1)
	{
		int currentSample = reservoir.id;

		int previousSample = previousReservoir[pixelIndex].id;

		// calculate current pHat
		float distanceToLight = length(lights[currentSample].position - hitPoint);
		float3 lightEmission = lights[currentSample].emission;
		float3 lightDirection = normalize(lights[currentSample].position - hitPoint);
		float cosTheta = dot(normalLocal, lightDirection);
		if (cosTheta < 0) cosTheta = 0.;
		float lightAttenuation = 1 / (distanceToLight * distanceToLight);
		float3 BRDF = obj->color * INV_PI * cosTheta;
		float3 lightIntensity = lightEmission * lightAttenuation;
		float3 radiance = BRDF * lightIntensity;
		float currPHat = length(radiance);

		// calculate previous pHat
		distanceToLight = length(lights[previousSample].position - hitPoint);
		lightEmission = lights[previousSample].emission;
		lightDirection = normalize(lights[previousSample].position - hitPoint);
		cosTheta = dot(normalLocal, lightDirection);
		if (cosTheta < 0) cosTheta = 0.;
		lightAttenuation = 1 / (distanceToLight * distanceToLight);
		BRDF = obj->color * INV_PI * cosTheta;
		lightIntensity = lightEmission * lightAttenuation;
		radiance = BRDF * lightIntensity;
		float prevPHat = length(radiance);

		// calculate MIS weights for both samples
		float currentMISWeight = currPHat / (20.0 * prevPHat + currPHat);
		float previousMISWeight = 20.0 * prevPHat / (20.0 * prevPHat + currPHat);

		// calculate the weight of the samples
		float currentSampleWeight = currPHat > 0.0 ? (1. / currPHat) * reservoir.wSum : 0.0;
		float previousSampleWeight = prevPHat > 0.0 ? (1. / prevPHat) * previousReservoir[pixelIndex].wSum : 0.0;

		// calculate the contribution weight of the samples
		float currentContributionWeight = currPHat * currentMISWeight * currentSampleWeight;
		float previousContributionWeight = prevPHat * previousMISWeight * previousSampleWeight;

		temporal_reservoir.addSample(currentSample, currentContributionWeight, randstate);
		temporal_reservoir.addSample(previousSample, previousContributionWeight, randstate);

		// Update the reservoir
		if (temporal_reservoir.wSum > 0.0)
		{
			reservoir.id = temporal_reservoir.id;
			reservoir.wSum = temporal_reservoir.wSum;
		}	
	}

	// Update the current and previous reservoir
	currentReservoir[pixelIndex] = reservoir;
	previousReservoir[pixelIndex] = reservoir;
}

__device__ float3 SpatialReuseAndFinalColor(const int &frameNumber, const Ray& r, const int& M, hiprandState *randstate, int2 index, int pixelIndex, 
							Reservoir *previousReservoir, Reservoir *currentReservoir,
							bool temporalReuse, bool spatialReuse)
{
	float t; // distance to intersection
	int id = 0; // id of intersected object
	Ray ray = r;

	if (!Intersect(ray, t, id)) return make_float3(0.0f, 0.0f, 0.0f);
	Sphere* obj = &spheres[id]; // the hit object
	float3 hitPoint = ray.origin + ray.direction * t;
	float3 normal = normalize(hitPoint - obj->position);
	float3 normalLocal = dot(normal, ray.direction) < 0 ? normal : normal * -1;
	bool specularReflection = false;

	// Perform specular reflection but not as iteration
	if (obj->refl == SPEC)
	{
		float3 reflectionDirection = ray.direction - normal * 2 * dot(normal, ray.direction);
		Ray reflectionRay = Ray(hitPoint + normal * 2e-2, reflectionDirection);
		if (!Intersect(reflectionRay, t, id)) return make_float3(0.0f, 0.0f, 0.0f);
		obj = &spheres[id]; // the hit object
		hitPoint = reflectionRay.origin + reflectionRay.direction * t;
		normal = normalize(hitPoint - obj->position);
		normalLocal = dot(normal, reflectionRay.direction) < 0 ? normal : normal * -1;
		specularReflection = true;
	}

	Reservoir reservoir;
	Reservoir spatial_reservoir;

	reservoir.normal = normalLocal;
	reservoir.depth = t;
	spatial_reservoir.normal = normalLocal;
	spatial_reservoir.depth = t;
	////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////// Spatial Reuse Pass /////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////////////////

	reservoir = currentReservoir[pixelIndex];

	if (spatialReuse && frameNumber > 1)
	{
		const uint k = 5; // total number of the spatial neighborhood
		uint finalSpatialNeighborCount = 5; // number of the spatial neighborhood that are similar
		uint range; // range of the spatial neighborhood
		if (!specularReflection)
		{
			range = 15;
		}
		else
		{
			range = 1;
		}
		int2 offset = make_int2(0, 0);
		int spatialSampleIndex[k];

		for (int i = 0; i < k; i ++)
		{
			offset.x = i > 0 ? int(hiprand_uniform(randstate) * 2.0 * range) - range : 0;
			offset.y = i > 0 ? int(hiprand_uniform(randstate) * 2.0 * range) - range : 0;

			int2 neighborIndex = max(make_int2(0, 0), min(index + offset, make_int2(scr_width - 1, scr_height - 1)));
			spatialSampleIndex[i] = (scr_height - neighborIndex.y - 1) * scr_width + neighborIndex.x;

			if (dot(reservoir.normal, currentReservoir[spatialSampleIndex[i]].normal) < 0.9 
				 && abs(reservoir.depth - currentReservoir[spatialSampleIndex[i]].depth) >= 0.1)
			{
				finalSpatialNeighborCount = finalSpatialNeighborCount - 1;
				spatialSampleIndex[i] = -1;
			}
		}

		// calculate MIS weight
		float spatialMISWeight = 1.f / (float)finalSpatialNeighborCount;

		for (int i = 0; i < k; i ++)
		{
			// skip dissimilar samples
			if (spatialSampleIndex[i] == -1)
			{
				continue;
			}

			int spatialSample = currentReservoir[spatialSampleIndex[i]].id;

			// calculate spatial pHat
			float distanceToLight = length(lights[spatialSample].position - hitPoint);
			float3 lightEmission = lights[spatialSample].emission;
			float3 lightDirection = normalize(lights[spatialSample].position - hitPoint);
			float cosTheta = dot(normalLocal, lightDirection);
			if (cosTheta < 0) cosTheta = 0.;
			// Check if the light is visible
			float visibility = 0;
			Ray shadowRay(hitPoint + normalLocal * 2e-2, lightDirection);
			if (!(Intersect(shadowRay, t, id) && t < distanceToLight - 2e-2))
			{
				visibility = 1;
			}
			float lightAttenuation = 1 / (distanceToLight * distanceToLight);
			float3 BRDF = obj->color * INV_PI * cosTheta;
			float3 lightIntensity = lightEmission * lightAttenuation;
			float3 radiance = BRDF * lightIntensity * visibility;
			float spatialPHat = length(radiance);

			// calculate the weight of the sample
			float spatialSampleWeight = spatialPHat > 0.0 ? (1. / spatialPHat) * currentReservoir[spatialSampleIndex[i]].wSum : 0.0;

			// calculate the contribution weight of the sample
			float spatialContributionWeight = spatialMISWeight * spatialPHat * spatialSampleWeight;

			// add sample to spatial reservoir
			spatial_reservoir.addSample(spatialSample, spatialContributionWeight, randstate);
		}
	}

	////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////// Final Color /////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////////////////

	// The chosen light to sample and the contribution weight
	int finalLight = spatialReuse ? spatial_reservoir.id : reservoir.id;

	// Sample the light
	float distanceToLight = length(lights[finalLight].position - hitPoint);
	float3 lightEmission = lights[finalLight].emission;
	float3 lightDirection = normalize(lights[finalLight].position - hitPoint);
	
	// Compute the Lambertian cosine
	float cosTheta = dot(normalLocal, lightDirection);
	if (cosTheta < 0) cosTheta = 0.;

	// Check if the light is visible
	float visibility = 0;
	Ray shadowRay(hitPoint + normalLocal * 2e-2, lightDirection);
	if (!(Intersect(shadowRay, t, id) && t < distanceToLight - 2e-2))
	{
		visibility = 1;
	}

	// Calculate the light attenuation
	float lightAttenuation = 1 / (distanceToLight * distanceToLight);

	// Compute the BRDF
	float3 BRDF = obj->color * INV_PI * cosTheta;

	// Compute the light intensity
	float3 lightIntensity = lightEmission * lightAttenuation;

	// Compute radiance
	float3 radiance = BRDF * lightIntensity * visibility;

	// Compute pHat
	float pHat = length(radiance);

	// pHat multiplied by the visibility
	float weight = pHat > 0.0 ? (1. / pHat) * (spatialReuse ? spatial_reservoir.wSum : reservoir.wSum) : 0.0;

	// Compute the direct illumination of Lambertian BRDF
	float3 color = radiance * weight;
	
	return color;
}

__device__ float3 DirectIllumination(const Ray& r, hiprandState *randstate)
{
	float t; // distance to intersection
	int id = 0; // id of intersected object
	Ray ray = r;
	
	if (!Intersect(ray, t, id)) return make_float3(0.0f, 0.0f, 0.0f);
	Sphere* obj = &spheres[id]; // the hit object
	float3 hitPoint = ray.origin + ray.direction * t;
	float3 normal = normalize(hitPoint - obj->position);
	float3 normalLocal = dot(normal, ray.direction) < 0 ? normal : normal * -1;

	// Perform specular reflection but not as iteration
	if (obj->refl == SPEC)
	{
		float3 reflectionDirection = ray.direction - normal * 2 * dot(normal, ray.direction);
		Ray reflectionRay = Ray(hitPoint + normal * 2e-2, reflectionDirection);
		if (!Intersect(reflectionRay, t, id)) return make_float3(0.0f, 0.0f, 0.0f);
		obj = &spheres[id]; // the hit object
		hitPoint = reflectionRay.origin + reflectionRay.direction * t;
		normal = normalize(hitPoint - obj->position);
		normalLocal = dot(normal, reflectionRay.direction) < 0 ? normal : normal * -1;
	}

	// Pick a random light from the scene to sample
	int lightsCount = sizeof(lights) / sizeof(PointLight);
	int randomLight = int(hiprand_uniform(randstate) * lightsCount);
	int lightToSample = randomLight > lightsCount - 1 ? lightsCount - 1 : randomLight;

	// Sample the light
	float distanceToLight = length(lights[lightToSample].position - hitPoint);
	float3 lightEmission = lights[lightToSample].emission;
	float3 lightDirection = normalize(lights[lightToSample].position - hitPoint);

	// Compute the Lambertian cosine
	float cosTheta = dot(normalLocal, lightDirection);
	if (cosTheta < 0) return make_float3(0.0f, 0.0f, 0.0f);

	// Check if the light is visible
	float visibility = 0;
	Ray shadowRay(hitPoint + normalLocal * 2e-2, lightDirection);
	if (!(Intersect(shadowRay, t, id) && t < distanceToLight - 2e-2))
	{
		visibility = 1;
	}

	// Calculate the inverse PDF of sampling the light
	float invPdf = (float)lightsCount;

	// Calculate the light attenuation
	float lightAttenuation = 1 / (distanceToLight * distanceToLight);

	// Compute the direct illumination of Lambertian BRDF
	float3 shadingColor = make_float3(0.0f, 0.0f, 0.0f);
	shadingColor += obj->color * lightEmission * INV_PI * cosTheta * lightAttenuation * visibility * invPdf;

	return shadingColor;
}

// union struct required for mapping pixel colours to OpenGL buffer
union Colour  // 4 bytes = 4 chars = 1 float
{
	float c;
	uchar4 components;
};

// hash function to calculate new seed for each frame
// see http://www.reedbeta.com/blog/2013/01/12/quick-and-easy-gpu-random-numbers-in-d3d11/
uint WangHash(uint a) {
	a = (a ^ 61) ^ (a >> 16);
	a = a + (a << 3);
	a = a ^ (a >> 4);
	a = a * 0x27d4eb2d;
	a = a ^ (a >> 15);
	return a;
}

__global__ void RISAndTemporalReuseKernel(float3 *finalOutputBuffer, int frameNumber, uint hashedFrameNumber, 
							  Reservoir *previousReservoir, Reservoir *currentReservoir, 
							  bool useReSTIR, bool temporalReuse, bool spatialReuse) {
	// assign a CUDA thread to every pixel (x,y) 
    // blockIdx, blockDim and threadIdx are CUDA specific keywords
    // replaces nested outer loops in CPU code looping over image rows and image columns 
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;   
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	int i = (scr_height - y - 1)*scr_width + x; // index of current pixel (calculated using thread index) 

	// global threadId, see richiesams blogspot
	int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	// create random number generator, see RichieSams blogspot
	hiprandState randState; // state of the random number generator, to prevent repetition
	hiprand_init(hashedFrameNumber + threadId, 0, 0, &randState);

	Ray cam(make_float3(50, 52, 295.6), normalize(make_float3(0, -0.042612, -1))); // first hardcoded camera ray(origin, direction)
	float3 cx = make_float3(scr_width * .5135 / scr_height, 0.0f, 0.0f); // ray direction offset in x direction
    float3 cy = normalize(cross(cx, cam.direction)) * .5135; // ray direction offset in y direction (.5135 is field of view angle)
        
	// compute primary ray direction
	float3 d = cam.direction + cx*((.25 + x) / scr_width - .5) + cy*((.25 + y) / scr_height - .5);
		
	// create primary ray, add incoming radiance to pixelcolor

	RISAndTemporalReuse(frameNumber, Ray(cam.origin + d * 40, normalize(d)), 32, &randState, make_int2(x, y), i, 
						  previousReservoir, currentReservoir, 
						  temporalReuse, spatialReuse); 
}

__global__ void SpatialReuseAndFinalColorKernel(float3 *finalOutputBuffer, int frameNumber, uint hashedFrameNumber, 
							  Reservoir *previousReservoir, Reservoir *currentReservoir, 
							  bool useReSTIR, bool temporalReuse, bool spatialReuse) {
	// assign a CUDA thread to every pixel (x,y) 
    // blockIdx, blockDim and threadIdx are CUDA specific keywords
    // replaces nested outer loops in CPU code looping over image rows and image columns 
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;   
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	int i = (scr_height - y - 1)*scr_width + x; // index of current pixel (calculated using thread index) 

	// global threadId, see richiesams blogspot
	int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	// create random number generator, see RichieSams blogspot
	hiprandState randState; // state of the random number generator, to prevent repetition
	hiprand_init(hashedFrameNumber + threadId, 0, 0, &randState);

	Ray cam(make_float3(50, 52, 295.6), normalize(make_float3(0, -0.042612, -1))); // first hardcoded camera ray(origin, direction)
	float3 cx = make_float3(scr_width * .5135 / scr_height, 0.0f, 0.0f); // ray direction offset in x direction
    float3 cy = normalize(cross(cx, cam.direction)) * .5135; // ray direction offset in y direction (.5135 is field of view angle)
    float3 r; // r is final pixel color     

	r = make_float3(0.0f); // reset r to zero for every pixel 
        
	// compute primary ray direction
	float3 d = cam.direction + cx*((.25 + x) / scr_width - .5) + cy*((.25 + y) / scr_height - .5);
		
	// create primary ray, add incoming radiance to pixelcolor
	r = r + SpatialReuseAndFinalColor(frameNumber, Ray(cam.origin + d * 40, normalize(d)), 32, &randState, make_int2(x, y), i, 
						  previousReservoir, currentReservoir, 
						  temporalReuse, spatialReuse); 

	Colour fcolour;
	float3 colour = make_float3(clamp(r.x, 0.0f, 1.0f), clamp(r.y, 0.0f, 1.0f), clamp(r.z, 0.0f, 1.0f));
	
	// convert from 96-bit to 24-bit colour + perform gamma correction
  	fcolour.components = make_uchar4((unsigned char)(powf(colour.x, 1 / 2.2f) * 255),
    (unsigned char)(powf(colour.y, 1 / 2.2f) * 255),
    (unsigned char)(powf(colour.z, 1 / 2.2f) * 255),1);

	finalOutputBuffer[i] = make_float3(x, y, fcolour.c);
}

__global__ void DIKernel(float3 *finalOutputBuffer, int frameNumber, uint hashedFrameNumber, 
							  Reservoir *previousReservoir, Reservoir *currentReservoir, 
							  bool useReSTIR, bool temporalReuse, bool spatialReuse) {
	// assign a CUDA thread to every pixel (x,y) 
    // blockIdx, blockDim and threadIdx are CUDA specific keywords
    // replaces nested outer loops in CPU code looping over image rows and image columns 
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;   
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	int i = (scr_height - y - 1)*scr_width + x; // index of current pixel (calculated using thread index) 

	// global threadId, see richiesams blogspot
	int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	// create random number generator, see RichieSams blogspot
	hiprandState randState; // state of the random number generator, to prevent repetition
	hiprand_init(hashedFrameNumber + threadId, 0, 0, &randState);

	Ray cam(make_float3(50, 52, 295.6), normalize(make_float3(0, -0.042612, -1))); // first hardcoded camera ray(origin, direction)
	float3 cx = make_float3(scr_width * .5135 / scr_height, 0.0f, 0.0f); // ray direction offset in x direction
    float3 cy = normalize(cross(cx, cam.direction)) * .5135; // ray direction offset in y direction (.5135 is field of view angle)
    float3 r; // r is final pixel color     

	r = make_float3(0.0f); // reset r to zero for every pixel 
        
	// compute primary ray direction
	float3 d = cam.direction + cx*((.25 + x) / scr_width - .5) + cy*((.25 + y) / scr_height - .5);
		
	// create primary ray, add incoming radiance to pixelcolor
	r = r + DirectIllumination(Ray(cam.origin + d * 40, normalize(d)), &randState);

	Colour fcolour;
	float3 colour = make_float3(clamp(r.x, 0.0f, 1.0f), clamp(r.y, 0.0f, 1.0f), clamp(r.z, 0.0f, 1.0f));
	
	// convert from 96-bit to 24-bit colour + perform gamma correction
  	fcolour.components = make_uchar4((unsigned char)(powf(colour.x, 1 / 2.2f) * 255),
    (unsigned char)(powf(colour.y, 1 / 2.2f) * 255),
    (unsigned char)(powf(colour.z, 1 / 2.2f) * 255),1);

	finalOutputBuffer[i] = make_float3(x, y, fcolour.c);
}

void RenderGate(float3* finalOutputBuffer, int frameNumber, uint hashedFrameNumber, 
				 Reservoir *previousReservoir, Reservoir *currentReservoir, 
				 bool useReSTIR, bool temporalReuse, bool spatialReuse) {
    // dim3 is CUDA specific type, block and grid are required to schedule CUDA threads over streaming multiprocessors
    dim3 block(16, 16, 1);   
    dim3 grid(scr_width / block.x, scr_height / block.y, 1);
    
    // schedule threads on device and launch CUDA kernel from host
	if (useReSTIR)
	{
		RISAndTemporalReuseKernel <<< grid, block >>>(finalOutputBuffer, frameNumber, hashedFrameNumber, 
									  previousReservoir, currentReservoir, 
									  useReSTIR, temporalReuse, spatialReuse);
		SpatialReuseAndFinalColorKernel <<< grid, block >>>(finalOutputBuffer, frameNumber, hashedFrameNumber, 
									  previousReservoir, currentReservoir, 
									  useReSTIR, temporalReuse, spatialReuse);
	}
	else
	{
    	DIKernel <<< grid, block >>>(finalOutputBuffer, frameNumber, hashedFrameNumber, 
									  previousReservoir, currentReservoir, 
									  useReSTIR, temporalReuse, spatialReuse);  
	}
	// Wait for GPU to finish before accessing on host
  	hipDeviceSynchronize();
}

__device__ static float getrandom(unsigned int *seed0, unsigned int *seed1) {
    *seed0 = 36969 * ((*seed0) & 65535) + ((*seed0) >> 16);  // hash the seeds using bitwise AND and bitshifts
    *seed1 = 18000 * ((*seed1) & 65535) + ((*seed1) >> 16);

    unsigned int ires = ((*seed0) << 16) + (*seed1);

    // Convert to float
    union {
    float f;
    unsigned int ui;
    } res;

    res.ui = (ires & 0x007fffff) | 0x40000000;  // bitwise AND, bitwise OR

    return (res.f - 2.f) / 2.f;
}

__device__ float3 Reference_DI(const Ray& r, unsigned int *s1, unsigned int *s2)
{
	float t; // distance to intersection
	int id = 0; // id of intersected object
	Ray ray = r;
	
	if (!Intersect(ray, t, id)) return make_float3(0.0f, 0.0f, 0.0f);
	Sphere* obj = &spheres[id]; // the hit object
	float3 hitPoint = ray.origin + ray.direction * t;
	float3 normal = normalize(hitPoint - obj->position);
	float3 normalLocal = dot(normal, ray.direction) < 0 ? normal : normal * -1;

	// Perform specular reflection but not as iteration
	if (obj->refl == SPEC)
	{
		float3 reflectionDirection = ray.direction - normal * 2 * dot(normal, ray.direction);
		Ray reflectionRay = Ray(hitPoint + normal * 2e-2, reflectionDirection);
		if (!Intersect(reflectionRay, t, id)) return make_float3(0.0f, 0.0f, 0.0f);
		obj = &spheres[id]; // the hit object
		hitPoint = reflectionRay.origin + reflectionRay.direction * t;
		normal = normalize(hitPoint - obj->position);
		normalLocal = dot(normal, reflectionRay.direction) < 0 ? normal : normal * -1;
	}

	// Pick a random light from the scene to sample
	int lightsCount = sizeof(lights) / sizeof(PointLight);
	int randomLight = int(getrandom(s1, s2) * lightsCount);
	int lightToSample = randomLight > lightsCount - 1 ? lightsCount - 1 : randomLight;

	// Sample the light
	float distanceToLight = length(lights[lightToSample].position - hitPoint);
	float3 lightEmission = lights[lightToSample].emission;
	float3 lightDirection = normalize(lights[lightToSample].position - hitPoint);

	// Compute the Lambertian cosine
	float cosTheta = dot(normalLocal, lightDirection);
	if (cosTheta < 0) return make_float3(0.0f, 0.0f, 0.0f);

	// Check if the light is visible
	float visibility = 0;
	Ray shadowRay(hitPoint + normalLocal * 2e-2, lightDirection);
	if (!(Intersect(shadowRay, t, id) && t < distanceToLight - 2e-2))
	{
		visibility = 1;
	}

	// Calculate the inverse PDF of sampling the light
	float invPdf = (float)lightsCount;

	// Calculate the light attenuation
	float lightAttenuation = 1 / (distanceToLight * distanceToLight);

	// Compute the direct illumination of Lambertian BRDF
	float3 shadingColor = make_float3(0.0f, 0.0f, 0.0f);
	shadingColor += obj->color * lightEmission * INV_PI * cosTheta * lightAttenuation * visibility * invPdf;

	return shadingColor;
}

__global__ void reference_kernel(float3 *output) {
	// assign a CUDA thread to every pixel (x,y) 
    // blockIdx, blockDim and threadIdx are CUDA specific keywords
    // replaces nested outer loops in CPU code looping over image rows and image columns 
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;   
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	int i = (scr_height - y - 1)*scr_width + x; // index of current pixel (calculated using thread index) 

	unsigned int s1 = x;
	unsigned int s2 = y;

	Ray cam(make_float3(50, 52, 295.6), normalize(make_float3(0, -0.042612, -1))); // first hardcoded camera ray(origin, direction)
	float3 cx = make_float3(scr_width * .5135 / scr_height, 0.0f, 0.0f); // ray direction offset in x direction
    float3 cy = normalize(cross(cx, cam.direction)) * .5135; // ray direction offset in y direction (.5135 is field of view angle)
    float3 r; // r is final pixel color     

	r = make_float3(0.0f); // reset r to zero for every pixel 
        
	// compute primary ray direction
	float3 d = cam.direction + cx*((.25 + x) / scr_width - .5) + cy*((.25 + y) / scr_height - .5);
		
	// create primary ray, add incoming radiance to pixelcolor
	for (int s = 0; s < 32768; s ++)
	{
		r = r + Reference_DI(Ray(cam.origin + d * 40, normalize(d)), &s1, &s2) / 32768.0;
	}

	float3 colour = make_float3(clamp(r.x, 0.0f, 1.0f), clamp(r.y, 0.0f, 1.0f), clamp(r.z, 0.0f, 1.0f));

	output[i] = colour;
}

void produceReference()
{
	float3* output_h = new float3[scr_width * scr_height]; // allocate memory for the image on the host
	float3* output_d; // allocate memory for the image on the device

	// allocate memory on the device
	hipMalloc(&output_d, scr_width * scr_height * sizeof(float3));

	dim3 block(16, 16, 1); // block dimensions
	dim3 grid(scr_width / block.x, scr_height / block.y, 1); // grid dimensions

	printf("Rendering reference image...\n");

	reference_kernel <<< grid, block >>>(output_d); // schedule threads on device and launch CUDA kernel from host

	hipMemcpy(output_h, output_d, scr_width * scr_height * sizeof(float3), hipMemcpyDeviceToHost); // copy the result back to the host

	hipFree(output_d); // free memory on the device

	printf("Reference image rendered.\n");

	// write the image to a file
	FILE *f = fopen("reference.ppm", "w");          
    fprintf(f, "P3\n%d %d\n%d\n", scr_width, scr_height, 255);
    for (int i = 0; i < scr_width*scr_height; i++)  // loop over pixels, write RGB values
    fprintf(f, "%d %d %d ", ToInt(output_h[i].x),
                            ToInt(output_h[i].y),
                            ToInt(output_h[i].z));

    printf("Saved image to 'reference.ppm'\n");

    delete[] output_h;
}